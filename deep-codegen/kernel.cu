#include "hip/hip_runtime.h"
#include "kernel.h"
#include <iostream>

__global__ void addVec(float *x, float *y, float *z)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    z[i] = x[i] + y[i];
}

#define TILE_WIDTH 32
__global__ void MatrixMulKernel(float *d_A, float *d_B, float *d_C, int M, int K, int N)
{
    // Tiled
    __shared__ float Ads[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Bds[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    float _P_x = 0.0;

    // t for tile index
    for (int t = 0; t < K; t += TILE_WIDTH)
    {
        if (row < M && t + tx < K)
            // memory coalescing: C1 + ty * K + tx
            Ads[ty][tx] = d_A[row * K + t + tx]; // A[row][t + tx];
        else
            Ads[ty][tx] = 0.0;

        if (col < N && t + ty < K)
            // memory coalescing: C2 + ty * N + tx
            Bds[ty][tx] = d_B[(t + ty) * N + col]; // B[t + ty][col];
        else
            Bds[ty][tx] = 0.0;

        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; ++k)
            _P_x += Ads[ty][k] * Bds[k][tx]; // avoid shared memory bank conflict

        __syncthreads();
    }
    if (row < M && col < N)
        d_C[row * N + col] = _P_x;
}

void addVec(array1d_t<float> &input1, array1d_t<float> &input2, array1d_t<float> &output)
{
    int N = input1.col_count;
    int threadsPerBlock = 32;
    int blocks = ceil(N / (float)threadsPerBlock);
    addVec<<<blocks, threadsPerBlock>>>(input1.data_ptr, input2.data_ptr, output.data_ptr);
}

void MatrixMuliplication(array2d_t<float> &input1, array2d_t<float> &input2, array2d_t<float> &output)
{
    int M = input1.row_count;
    int K = input1.col_count;
    int N = input2.col_count;
    // Throw an error or handle the dimension mismatch case
    if (input2.row_count != K || output.row_count != M || output.col_count != N)
    {
        std::cerr << "Error: Matrix dimensions are not compatible for multiplication." << std::endl;
        return;
    }

    dim3 dimGrid(ceil(N / (float)TILE_WIDTH), ceil(M / (float)TILE_WIDTH), 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    MatrixMulKernel<<<dimGrid, dimBlock>>>(input1.data_ptr, input2.data_ptr, output.data_ptr, M, K, N);
}